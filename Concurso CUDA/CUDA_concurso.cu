/*
 * Probabilistic approach to locate maximum heights
 * Hill Climbing + Montecarlo
 *
 	Código realizado por Héctor Toribio González y Juan Torres Viloria
 
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2021/2022
 *
 * v1.1
 *
 * (c) 2022 Arturo Gonzalez Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>







/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


#define	PRECISION	10000

/*
 * Structure to represent a climbing searcher
 * 	This structure can be changed and/or optimized by the students
 */
typedef struct {
	int id;				// Searcher identifier
	int pos_row, pos_col;		// Position in the grid
	int steps;			// Steps count
	int follows;			// When it finds an explored trail, who searched that trail
} Searcher;




/*
 * Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Macro function to simplify accessing with two coordinates to a flattened array
 * 	This macro-function can be changed and/or optimized by the students
 */
#define accessMat( arr, exp1, exp2 )	arr[ (int)(exp1) * columns + (int)(exp2) ]


/*
 * Function: Generate height for a given position
 * 	This function can be changed and/or optimized by the students
 */
__device__ int get_height( int x, int y, int rows, int columns, float x_min, float x_max, float y_min, float y_max  ) {
	/* Calculate the coordinates of the point in the ranges */
	float x_coord = x_min + ( (x_max - x_min) / rows ) * x;
	float y_coord = y_min + ( (y_max - y_min) / columns ) * y;
	/* Compute function value */
	float value = 2 * sin(x_coord) * cos(y_coord/2) + log( fabs(y_coord - M_PI_2) );
	/* Transform to fixed point precision */
	int fixed_point = (int)( PRECISION * value );
	return fixed_point;
}

/*
 * Function: Climbing step
 * 	This function can be changed and/or optimized by the students
 */
__device__ int climbing_step( int rows, int columns, Searcher *searchers, int search, int *heights, int *trails, int *tainted, float x_min, float x_max, float y_min, float y_max ) {
	int search_flag = 0;

	/* Annotate one step more, landing counts as the first step */
	searchers[ search ].steps ++;

	/* Get starting position */
	int pos_row = searchers[ search ].pos_row;
	int pos_col = searchers[ search ].pos_col;

	/* Stop if searcher finds another trail */
	int check;

	check = atomicCAS(&accessMat( tainted, pos_row, pos_col ), 0, 1);

	if ( check != 0 ) {
		search_flag = 1;
	}
	else {
		/* Annotate the trail */
		accessMat( trails, pos_row, pos_col ) = search;

		/* Compute the height */
		accessMat( heights, pos_row, pos_col ) = get_height( pos_row, pos_col, rows, columns, x_min, x_max, y_min, y_max );

		/* Locate the highest climbing direction */
		float local_max = accessMat( heights, pos_row, pos_col );
		int climbing_direction = 0;
		if ( pos_row > 0 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row-1, pos_col ) == INT_MIN )
				accessMat( heights, pos_row-1, pos_col ) = get_height( pos_row-1, pos_col, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row-1, pos_col ) > local_max ) {
				climbing_direction = 1;
				local_max = accessMat( heights, pos_row-1, pos_col );
			}
		}
		if ( pos_row < rows-1 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row+1, pos_col ) == INT_MIN )
				accessMat( heights, pos_row+1, pos_col ) = get_height( pos_row+1, pos_col, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row+1, pos_col ) > local_max ) {
				climbing_direction = 2;
				local_max = accessMat( heights, pos_row+1, pos_col );
			}
		}
		if ( pos_col > 0 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row, pos_col-1 ) == INT_MIN )
				accessMat( heights, pos_row, pos_col-1 ) = get_height( pos_row, pos_col-1, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row, pos_col-1 ) > local_max ) {
				climbing_direction = 3;
				local_max = accessMat( heights, pos_row, pos_col-1 );
			}
		}
		if ( pos_col < columns-1 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row, pos_col+1 ) == INT_MIN )
				accessMat( heights, pos_row, pos_col+1 ) = get_height( pos_row, pos_col+1, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row, pos_col+1 ) > local_max ) {
				climbing_direction = 4;
				local_max = accessMat( heights, pos_row, pos_col+1 );
			}
		}

		/* Stop if local maximum is reached */
		if ( climbing_direction == 0 ) {
			searchers[ search ].follows = search;
			search_flag = 1;
		}

		/* Move in the chosen direction: 0 does not change coordinates */
		switch( climbing_direction ) {
			case 1: pos_row--; break;
			case 2: pos_row++; break;
			case 3: pos_col--; break;
			case 4: pos_col++; break;
		}
		searchers[ search ].pos_row = pos_row;
		searchers[ search ].pos_col = pos_col;
	}

	/* Return a flag to indicate if search should stop */
	return search_flag;
}


#ifdef DEBUG
/*
 * Function: Print the current state of the simulation
 */
void print_heights( int rows, int columns, int *heights ) {
	/*
	 * You don't need to optimize this function, it is only for pretty
	 * printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;
	printf("Heights:\n");
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( heights, i, j ) != INT_MIN )
				printf(" %6d", accessMat( heights, i, j ) );
			else
				printf("       ");
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n\n");
}

void print_trails( int rows, int columns, int *trails ) {
	/*
	 * You don't need to optimize this function, it is only for pretty
	 * printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;
	printf("Trails:\n");
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( trails, i, j ) != -1 )
				printf("%7d", accessMat( trails, i, j ) );
			else
				printf("       ", accessMat( trails, i, j ) );
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n\n");
}
#endif // DEBUG

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<rows> <columns> <x_min> <x_max> <y_min> <y_max> <searchers_density> <short_rnd1> <short_rnd2> <short_rnd3>\n");
	fprintf(stderr,"\n");
}


//KERNEL PARA INICIALIZAR LOS BUSCADORES (bucle 3.3)
__global__ void  kernel_InitSearchers(Searcher *searchersDevice, int *total_stepsDevice, int num_searchers){

	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

	if ( globalPos < num_searchers ) {
		//searchersDevice[ globalPos ].id = globalPos;
		searchersDevice[ globalPos ].steps = 0;
		searchersDevice[ globalPos ].follows = -1;
		total_stepsDevice[ globalPos ] = 0;
	}
}

//KERNEL PARA INICIALIZAR EL TERRENO. (bucle 3.2)
/*
__global__ void kernel_InitTerrain(Searcher *searchersDevice, int *heightsDevice, int *trailsDevice, int *taintedDevice){
	arr[ (int)(exp1) * columns + (int)(exp2) ]
}
*/

//KERNEL PARA LA FUNCION CLIMBING_STEP (bucle 4)
__global__ void  kernel_climbStep(Searcher *searchersDevice, int *taintedDevice, int *heightsDevice,int *trailsDevice, int rows, int columns, float x_min, float x_max, float y_min, float y_max, int num_searchers){
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("posicion global: %d ", globalPos);
	int search_flag = 0;
	if(globalPos<num_searchers){
			while( ! search_flag ) {
				search_flag = climbing_step( rows, columns, searchersDevice, globalPos, heightsDevice, trailsDevice, taintedDevice, x_min, x_max, y_min, y_max );
			}
	}

}



//KERNEL PARA FOLLOWERS (bucle5)
__global__ void kernel_ComputeFollowers(Searcher *searchersDevice, int num_searchers){

	int globalPos = threadIdx.x + (blockIdx.x * blockDim.x);

	if ( globalPos < num_searchers ) {
		int search_flag = 0;
		int parent = globalPos;
		int follows_to = searchersDevice[ parent ].follows;
		while( ! search_flag ) {
			if ( follows_to == parent ) search_flag = 1;
			else {
				parent = follows_to;
				follows_to = searchersDevice[ parent ].follows;
			}
		}
		searchersDevice[ globalPos ].follows = follows_to;
	}
}

//KERNEL PARA STEPS (bucle 6)
__global__ void  kernel_Trail(Searcher *searchersDevice, int *total_stepsDevice, int num_searchers){
	int globalPos = threadIdx.x + (blockIdx.x * blockDim.x);

	if(globalPos<num_searchers){
		int pos_max = searchersDevice[ globalPos ].follows;
		atomicAdd(&total_stepsDevice[ pos_max ], searchersDevice[ globalPos ].steps);
	}
}


__global__ void  kernel_condicionCarrera(Searcher *searchersDevice, int *trailsDevice, int num_searchers, int columns){
	int globalPos = threadIdx.x + (blockIdx.x * blockDim.x);

	if(globalPos<num_searchers){
		searchersDevice[ globalPos ].follows = accessMat( trailsDevice, searchersDevice[ globalPos].pos_row, searchersDevice[ globalPos ].pos_col );
	}
}

__global__ void  kernel_SumaTainted (int *tainted, int num_searchers, int *suma){

	// Compute the global position of the thread in the grid
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

	// Shared memory: One element per thread in the block
	// Call this kernel with the proper third launching parameter
	extern __shared__ int buffer[ ];

	// Load array values in the shared memory (0 if out of the array)
	if ( globalPos < num_searchers ) {
		buffer[ threadIdx.x ] = tainted[ globalPos ];
	}
	else buffer[ threadIdx.x ] = 0;

	// Wait for all the threads of the block to finish
	__syncthreads();

	// Reduction tree
	for( int step=blockDim.x/2; step>=1; step /= 2 ) {
		if ( threadIdx.x < step )
				buffer[ threadIdx.x ] += buffer[ threadIdx.x + step ];
		__syncthreads();
	}

	// The maximum value of this block is on the first position of buffer
	if ( threadIdx.x == 0 )

		atomicAdd( suma, buffer[0] );
}

__global__ void kernel_SumaHeights(int* array, int size, unsigned long long int *result){
    // Compute the global position of the thread in the grid
    int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

    // Shared memory: One element per thread in the block
    // Call this kernel with the proper third launching parameter
    extern __shared__ int buffer[ ];

    // Load array values in the shared memory (0 if out of the array)
    if ( globalPos < size) { //and array[globalPos] != INT_MIN
        if( array[globalPos] != INT_MIN ) {
            buffer[ threadIdx.x ] = array[ globalPos ];
        } else {
            buffer[ threadIdx.x ] = 0;
        }
    }
    else buffer[ threadIdx.x ] = 0;

    // Wait for all the threads of the block to finish
    __syncthreads();

    // Reduction tree
    for( int step=blockDim.x/2; step>=1; step /= 2 ) {
        if ( threadIdx.x < step )
            buffer[ threadIdx.x ] += buffer[ threadIdx.x + step ];
        __syncthreads();
    }

    // The maximum value of this block is on the first position of buffer
    if ( threadIdx.x == 0 )
        atomicAdd( result, buffer[0] );
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	// This eliminates the buffer of stdout, forcing the messages to be printed immediately
	setbuf(stdout,NULL);

	int i,j, search;



	// Simulation data
	int rows, columns;		// Matrix sizes
	float x_min, x_max;		// Limits of the terrain x coordinates
	float y_min, y_max;		// Limits of the terrain y coordinates

	float searchers_density;	// Density of hill climbing searchers
	unsigned short random_seq[3];	// Status of the random sequence

	int *heights;			// Heights of the terrain points
	int *trails;			// Searchers trace and trails
	int *tainted;			// Position found in a search
	int num_searchers;		// Number of searchers
	Searcher *searchers;		// Searchers data
	int *total_steps;		// Annotate accumulated steps to local maximums

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc != 11) {
		fprintf(stderr, "-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	rows = atoi( argv[1] );
	columns = atoi( argv[2] );
	x_min = atof( argv[3] );
	x_max = atof( argv[4] );
	y_min = atof( argv[5] );
	y_max = atof( argv[6] );
	searchers_density = atof( argv[7] );

	/* 1.3. Read random sequences initializer */
	for( i=0; i<3; i++ ) {
		random_seq[i] = (unsigned short)atoi( argv[8+i] );
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d\n", rows, columns);
	printf("Arguments, x_range: ( %d, %d ), y_range( %d, %d )\n", x_min, x_max, y_min, y_max );
	printf("Arguments, searchers_density: %f\n", searchers_density );
	printf("Arguments, Init Random Sequence: %hu,%hu,%hu\n", random_seq[0], random_seq[1], random_seq[2]);
	printf("\n");
#endif // DEBUG



	/* 2. Start global timer */
	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

	/* 3. Initialization */
	/* 3.1. Memory allocation */

	num_searchers = (int)( rows * columns * searchers_density );


	int NHILOS = 1024;
	int NHILOS_REDUCTION = 1024;
	int NBLOQUES = num_searchers/NHILOS;
	int NBLOQUES_REDUCTION = (rows *columns) / NHILOS_REDUCTION;

	if(num_searchers % NHILOS != 0){
		NBLOQUES ++;
	}
	
	if((rows *columns) % NHILOS_REDUCTION != 0){
		NBLOQUES_REDUCTION ++;
	}




	searchers = (Searcher *)malloc( sizeof(Searcher) * num_searchers );
	total_steps = (int *)malloc( sizeof(int) * num_searchers );
	heights = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	trails = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	tainted = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );

	Searcher *searchersDevice;
	int *trailsDevice;
	int *total_stepsDevice;
	int *heightsDevice;
	int *taintedDevice;


	hipMalloc( (void**) &searchersDevice, sizeof(Searcher) * num_searchers);CHECK_CUDA_LAST();
	hipMalloc( (void**) &total_stepsDevice, sizeof(int) * num_searchers);CHECK_CUDA_LAST();
	hipMalloc( (void**) &trailsDevice, sizeof(int) * (size_t)rows * (size_t)columns );CHECK_CUDA_LAST();
	hipMalloc( (void**) &heightsDevice, sizeof(int) * (size_t)rows * (size_t)columns );CHECK_CUDA_LAST();
	hipMalloc( (void**) &taintedDevice, sizeof(int) * (size_t)rows * (size_t)columns );CHECK_CUDA_LAST();

	if ( searchers == NULL || total_steps == NULL ) {
		fprintf(stderr,"-- Error allocating searchers structures for size: %d\n", num_searchers );
		exit( EXIT_FAILURE );
	}


	if ( heights == NULL || trails == NULL || tainted == NULL ) {
		fprintf(stderr,"-- Error allocating terrain structures for size: %d x %d \n", rows, columns );
		exit( EXIT_FAILURE );
	}


	/* 3.2. Terrain initialization */
	for( i=0; i<rows; i++ ) {
		for( j=0; j<columns; j++ ) {
			accessMat( heights, i, j ) = INT_MIN;
			accessMat( trails, i, j ) = -1;
			accessMat( tainted, i, j ) = 0;
		}
	}


	for( search = 0; search < num_searchers; search++ ) {
		searchers[ search ].pos_row = (int)( rows * erand48( random_seq ) );
		searchers[ search ].pos_col = (int)( columns * erand48( random_seq ) );
	}
	hipMemcpy (taintedDevice, tainted, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy (trailsDevice, trails, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy (heightsDevice, heights, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy (searchersDevice, searchers, sizeof(Searcher) * num_searchers, hipMemcpyHostToDevice);
	hipMemcpy (total_stepsDevice, total_steps, sizeof(int) * num_searchers, hipMemcpyHostToDevice);
	/* 3.3. Searchers initialization */
	kernel_InitSearchers<<<NBLOQUES, NHILOS>>>(searchersDevice, total_stepsDevice, num_searchers);CHECK_CUDA_LAST();


	/* 4. Compute searchers climbing trails */
	kernel_climbStep<<<NBLOQUES, NHILOS>>>(searchersDevice, taintedDevice, heightsDevice,trailsDevice, rows, columns, x_min, x_max, y_min, y_max, num_searchers);CHECK_CUDA_LAST();


	kernel_condicionCarrera<<<NBLOQUES, NHILOS>>>(searchersDevice, trailsDevice, num_searchers, columns);CHECK_CUDA_LAST();


	/* 5. Compute the leading follower of each searcher */
	kernel_ComputeFollowers<<<NBLOQUES, NHILOS>>>(searchersDevice, num_searchers);CHECK_CUDA_LAST();

	/* 6. Compute accumulated trail steps to each maximum */

	kernel_Trail<<<NBLOQUES, NHILOS>>>(searchersDevice, total_stepsDevice, num_searchers);CHECK_CUDA_LAST();
	hipMemcpy (total_steps, total_stepsDevice, sizeof(int) * num_searchers, hipMemcpyDeviceToHost);
	hipMemcpy (searchers, searchersDevice, sizeof(Searcher) * num_searchers, hipMemcpyDeviceToHost);
	//cudaMemcpy (tainted, taintedDevice, sizeof(int) * (size_t)rows * (size_t)columns, cudaMemcpyDeviceToHost);
	hipMemcpy (heights, heightsDevice, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);
	hipMemcpy (trails, trailsDevice, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);

	/* 7. Compute statistical data */
	int num_local_max = 0;
	int max_height = INT_MIN;
	int max_accum_steps = INT_MIN;
	int *total_tainted;
	unsigned long long int total_heights = 0;

	for( search = 0; search < num_searchers; search++ ) {
		/* Maximum of accumulated trail steps to a local maximum */
		if ( max_accum_steps < total_steps[ search ] )
			max_accum_steps = total_steps[ search ];

		/* If this searcher found a maximum, check the maximum value */
		if ( searchers[ search ].follows == search ) {
			num_local_max++;
			int pos_row = searchers[ search ].pos_row;
			int pos_col = searchers[ search ].pos_col;
			if ( max_height < accessMat( heights, pos_row, pos_col ) )
				max_height = accessMat( heights, pos_row, pos_col );
		}
	}


	int *sumaTainted;
	unsigned long long int *sumaHeights;
	
	//Memoria compartida
	//int compartida = NHILOS* sizeof(int);
	int compartida_reduction = NHILOS_REDUCTION *sizeof(int);
	
	hipMalloc( &sumaTainted, sizeof(int));CHECK_CUDA_LAST();
	hipMemsetAsync(sumaTainted, 0, sizeof(int));CHECK_CUDA_LAST();
	
	kernel_SumaTainted<<<NBLOQUES_REDUCTION, NHILOS_REDUCTION, compartida_reduction>>>(taintedDevice, rows*columns, sumaTainted);CHECK_CUDA_LAST();
	hipMemcpy(&total_tainted, sumaTainted, sizeof(int), hipMemcpyDeviceToHost);CHECK_CUDA_LAST();

	hipMalloc(&sumaHeights, sizeof(unsigned long long int));CHECK_CUDA_LAST();
	hipMemsetAsync(sumaHeights, 0, sizeof(unsigned long long int));CHECK_CUDA_LAST();
	
	kernel_SumaHeights<<<NBLOQUES_REDUCTION, NHILOS_REDUCTION, compartida_reduction>>>(heightsDevice, rows*columns, sumaHeights);CHECK_CUDA_LAST();
	hipMemcpy(&total_heights, sumaHeights, sizeof(unsigned long long int), hipMemcpyDeviceToHost);CHECK_CUDA_LAST();

	hipFree(searchersDevice);
	hipFree(total_stepsDevice);
	hipFree(trailsDevice);
	hipFree(heightsDevice);
	hipFree(taintedDevice);





/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 6.2. Results: Statistics */
	printf("Result: %d, %d, %d, %d, %llu\n\n",
			num_local_max,
			max_height,
			max_accum_steps,
			total_tainted,
			total_heights );

	/* 7. Free resources */
	free( searchers );
	free( total_steps );
	free( heights );
	free( trails );
	free( tainted );

	/* 8. End */
	return 0;
}
